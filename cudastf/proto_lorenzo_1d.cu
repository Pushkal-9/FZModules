#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstddef>
#include <stdexcept>

using namespace cuda::experimental::stf;


__global__ void kernel_lorenzo_1d(slice<float> input,
                                  size_t input_size,
                                  slice<uint16_t> quant_codes,
                                  slice<float> outlier_vals,
                                  slice<uint32_t> outlier_idxs,
                                  slice<uint32_t> outlier_num,
                                  double ebx2,
                                  double ebx2_r,
                                  size_t radius = 512) 
{
  __shared__ float buf[256];

  auto id = threadIdx.x + blockIdx.x * 256;
  auto data = [&](auto dx) -> float& { return buf[threadIdx.x + dx]; };

  // prequant
  if (id < input_size) { data(0) = round(input(id) * ebx2_r); }
  __syncthreads();

  // quantization
  float delta = data(0) - (threadIdx.x == 0 ? 0 : data(-1));
  bool quantizable = fabs(delta) < radius;
  float candidate = delta + radius;
  if (id < input_size) {
    quant_codes[id] = quantizable * static_cast<uint16_t>(candidate);
    if (not quantizable) {
      auto curr_idx = atomicAdd(outlier_num.data_handle(), 1);
      outlier_idxs(curr_idx) = id;
      outlier_vals(curr_idx) = candidate;
    }
  }
}